#include "hip/hip_runtime.h"
/*
 *  davinci_fwd_kinematics.cu
 *  Copyright (C) 2017  Wyatt S. Newman, Russell C. Jackson, and Tom Shkurti.
 *  Copyright (C) 2021 Ethan Shafer.
 *  
 *  This program is free software: you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation, either version 3 of the License, or
 *  (at your option) any later version.
 *  
 *  This program is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *  
 *  You should have received a copy of the GNU General Public License
 *  along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <davinci_kinematics_cuda/davinci_fwd_kinematics.cuh>

namespace davinci_kinematics_cuda {
	
	// use member fncs to compute and multiply successive transforms
	__host__ __device__
	Forward::Forward() {
		// affine describing frame0 w/rt base frame
		Eigen::Matrix3d R_0_wrt_base;
		Eigen::Vector3d Origin_0_wrt_base;
		Origin_0_wrt_base << 0, 0, 0;

		// choose x0 to point down, so will not have a joint-angle offset for pitch
		Eigen::Vector3d x_axis(0, 0, -1);
		// consistent triad
		Eigen::Vector3d y_axis(1, 0, 0);
		// points IN, so + rotation is consistent leaning to the robot's left
		Eigen::Vector3d z_axis(0, -1, 0);

		// R_{0/base} = [0  1  0
		//               0  0 -1
		//              -1  0  0 ]
		R_0_wrt_base.col(0) = x_axis;
		R_0_wrt_base.col(1) = y_axis;
		R_0_wrt_base.col(2) = z_axis;

		this->affine_frame0_wrt_base_.linear() = R_0_wrt_base;
		this->affine_frame0_wrt_base_.translation() = Origin_0_wrt_base;

		// fill in a static tool transform from frame6 to a frame of interest on the gripper
		set_gripper_jaw_length(davinci_kinematics_cuda::gripper_jaw_length);

		for (int i = 0; i < 7; i++) {
			this->theta_DH_offsets_[i] = davinci_kinematics_cuda::DH_q_offsets[i];

		}
		// don't put prismatic displacement here
		this->theta_DH_offsets_[2] = 0.0;
		
		
		this->dval_DH_offsets_ << 0, 0, davinci_kinematics_cuda::DH_q_offsets[2], 0, 0, 0, 0; // RN 20180713 EXPERIMENTAL

		resetDhGenericParams();
	}
	
	__host__ __device__
	void Forward::resetDhGenericParams() {
		for (int i = 0; i < 7; i++) {
			this->theta_DH_offsets_generic_[i] = davinci_kinematics_cuda::DH_q_offsets[i];
			this->DH_alpha_params_generic_[i] = davinci_kinematics_cuda::DH_alpha_params[i];
			this->DH_a_params_generic_[i] = davinci_kinematics_cuda::DH_a_params[i];
			if (i == 2) {
				this->theta_DH_offsets_generic_[i] = 0;
			}
		}

		this->dval_DH_offsets_generic_ << 0, 0, davinci_kinematics_cuda::DH_q_offsets[2], 0, 0, 0, 0;
		this->j1_scale_factor_generic_ = 1.0;
		this->j2_scale_factor_generic_ = 1.0;
		this->j3_scale_factor_generic_ = 1.0;
	}

	// given a vector of joint states in DaVinci coords, convert these into
	// equivalent DH parameters, theta and d
	__host__ __device__
	void Forward::convert_qvec_to_DH_vecs(const Vector7 &q_vec, Vector7 &thetas_DH_vec, Vector7 &dvals_DH_vec) {
		// +? -?
		thetas_DH_vec = this->theta_DH_offsets_;
		for (int i = 0; i < 7; i++) {
			// skip the linear joint.
			if (i == 2)
				continue;
			thetas_DH_vec(i) += q_vec(i);
		}

		dvals_DH_vec = this->dval_DH_offsets_;
		dvals_DH_vec(2) += q_vec(2); // RN original
	}

	__host__ __device__
	double Forward::dh_var_to_qvec(double dh_val, int index) {
		return (dh_val - davinci_kinematics_cuda::DH_q_offsets[index]);
	}

	// given 4 DH parameters, compute the corresponding transform as an affine3d
	__host__ __device__
	Eigen::Affine3d Forward::computeAffineOfDH(double a, double d, double alpha, double theta) {
		Eigen::Affine3d affine_DH;
		Eigen::Matrix3d R;
		Eigen::Vector3d p;

		double cq = cos(theta);
		double sq = sin(theta);
		double sa = sin(alpha);
		double ca = cos(alpha);
		R(0, 0) = cq;

		// - sin(q(i))*cos(alpha);
		R(0, 1) = -sq * ca;

		// sin(q(i))*sin(alpha);
		R(0, 2) = sq * sa;
		R(1, 0) = sq;
		// cos(q(i))*cos(alpha);
		R(1, 1) = cq * ca;
		R(1, 2) = -cq * sa;

		R(2, 0) = 0;
		R(2, 1) = sa;
		R(2, 2) = ca;
		affine_DH.linear() = R;

		p(0) = a * cq;
		p(1) = a * sq;
		p(2) = d;
		affine_DH.translation() = p;

		return affine_DH;
	}

	// provide DH theta and d values, return affine pose of gripper tip w/rt base frame
	// also computes all intermediate affine frames, w/rt base frame
	__host__ __device__
	void Forward::fwd_kin_solve_DH(const Vector7 &theta_vec, const Vector7 &d_vec, const unsigned int joint,
			Eigen::Affine3d &result) {
		// use or affect these member variables:
		Eigen::Affine3d affines_i_wrt_iminus1[7];
		Eigen::Affine3d affines_products[7];
		Eigen::Affine3d transform;
		
		double a, d, theta, alpha;

		for (int i = 0; i < 7; i++) {
			a = DH_a_params[i];
			d = d_vec(i);
			alpha = DH_alpha_params[i];
			theta = theta_vec(i);
			transform = computeAffineOfDH(a, d, alpha, theta);
			affines_i_wrt_iminus1[i] = transform;
		}
		
		affines_products[0] = this->affine_frame0_wrt_base_ * affines_i_wrt_iminus1[0];
		// RN Note that it starts from 1.
		for (int i = 1; i <= joint; i++) {
			affines_products[i] = affines_products[i - 1] * affines_i_wrt_iminus1[i];
		}
		result = affines_products[joint];
	}
	
	__host__ __device__
	Eigen::Affine3d Forward::fwd_kin_solve(const Vector7 &q_vec, const unsigned int desired_joint) {
		unsigned int joint = desired_joint - 1;
		Vector7 thetas_DH_vec;
		Vector7 dvals_DH_vec;
		convert_qvec_to_DH_vecs(q_vec, thetas_DH_vec, dvals_DH_vec);
		
		Eigen::Affine3d result;
		fwd_kin_solve_DH(thetas_DH_vec, dvals_DH_vec, joint, result);
		return result;
	}

	__host__ __device__
	Eigen::Affine3d Forward::fwd_kin_solve(const double *q_vec, const unsigned int desired_joint) {
		Vector7 q;
		q[0] = q_vec[0];
		q[1] = q_vec[1];
		q[2] = q_vec[2];
		q[3] = q_vec[3];
		q[4] = q_vec[4];
		q[5] = q_vec[5];
		q[6] = q_vec[6];
		return this->fwd_kin_solve(q, desired_joint);
	}

	__host__ __device__
	void Forward::set_gripper_jaw_length(double jaw_length) {
		this->gripper_jaw_length_ = jaw_length;
		this->affine_gripper_wrt_frame6_ = computeAffineOfDH(0, this->gripper_jaw_length_, 0, -davinci_kinematics_cuda::PI / 2);
	}

}  // namespace davinci_kinematics
