#include "hip/hip_runtime.h"
/*
 *  davinci_fwd_kinematics.cu
 *  Copyright (C) 2017  Wyatt S. Newman, Russell C. Jackson, and Tom Shkurti.
 *  Copyright (C) 2021 Ethan Shafer.
 *  
 *  This program is free software: you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation, either version 3 of the License, or
 *  (at your option) any later version.
 *  
 *  This program is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *  
 *  You should have received a copy of the GNU General Public License
 *  along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <davinci_kinematics_cuda/davinci_fwd_kinematics.cuh>

namespace davinci_kinematics_cuda {
	
	// use member fncs to compute and multiply successive transforms
	__host__ __device__
	Forward::Forward() {
		// affine describing frame0 w/rt base frame
		Eigen::Matrix3d R_0_wrt_base;
		Eigen::Vector3d Origin_0_wrt_base;
		Origin_0_wrt_base << 0, 0, 0;

		// choose x0 to point down, so will not have a joint-angle offset for pitch
		Eigen::Vector3d x_axis(0, 0, -1);
		// consistent triad
		Eigen::Vector3d y_axis(1, 0, 0);
		// points IN, so + rotation is consistent leaning to the robot's left
		Eigen::Vector3d z_axis(0, -1, 0);

		// R_{0/base} = [0  1  0
		//               0  0 -1
		//              -1  0  0 ]
		R_0_wrt_base.col(0) = x_axis;
		R_0_wrt_base.col(1) = y_axis;
		R_0_wrt_base.col(2) = z_axis;

		this->affine_frame0_wrt_base_.linear() = R_0_wrt_base;
		this->affine_frame0_wrt_base_.translation() = Origin_0_wrt_base;

		// fill in a static tool transform from frame6 to a frame of interest on the gripper
		set_gripper_jaw_length(davinci_kinematics_cuda::gripper_jaw_length);

		this->theta_DH_offsets_.resize(7);
		for (int i = 0; i < 7; i++) {
			this->theta_DH_offsets_(i) = davinci_kinematics_cuda::DH_q_offsets[i];

		}
		// don't put prismatic displacement here
		this->theta_DH_offsets_(2) = 0.0;

		this->dval_DH_offsets_.resize(7);
		// dval_DH_offsets_<< 0, 0 , DH_q_offsets[2], 0, 0, 0, 0;

		//dval_DH_offsets_<< 0, 0, 0, 0, 0, 0, 0; // RN 20180712A1 PSM1
		this->dval_DH_offsets_ << 0, 0, davinci_kinematics_cuda::DH_q_offsets[2], 0, 0, 0, 0; // RN 20180713 EXPERIMENTAL

		// resize MatrixXd Jacobian_ and initialize terms to 0's
		this->Jacobian_ = Eigen::MatrixXd::Zero(6, 6);

		resetDhGenericParams();
	}

	// fnc to extract a joint value from a JointState message;
	// provide the name of interest, as a C++ string, and provide the entire
	// jointState message;  will set the value of "qval" arg, if possible;
	// will return "true" or "false" to indicate if name was found on list
	/*bool Forward::get_jnt_val_by_name(std::string jnt_name, sensor_msgs::JointState jointState, double &qval) {
		//TODO replace sensor_msgs::JointState with appropriate data type

	}*/

	// given a vector of joint states in DaVinci coords, convert these into
	// equivalent DH parameters, theta and d
	__host__ __device__
	void Forward::convert_qvec_to_DH_vecs(const Vectorq7x1 &q_vec, Eigen::VectorXd &thetas_DH_vec, Eigen::VectorXd &dvals_DH_vec) {
		thetas_DH_vec.resize(7);
		// +? -?
		thetas_DH_vec = this->theta_DH_offsets_;
		for (int i = 0; i < 7; i++) {
			// skip the linear joint.
			if (i == 2)
				continue;
			thetas_DH_vec(i) += q_vec(i);
		}

		dvals_DH_vec.resize(7);
		dvals_DH_vec = this->dval_DH_offsets_;
		dvals_DH_vec(2) += q_vec(2); // RN original
	}

	__host__ __device__
	int Forward::check_jnts(const Vectorq7x1 &q_vec) {
		int result(0);
		for (int i(0); i < 6; i++) {
			if (q_vec(i) < davinci_kinematics_cuda::q_lower_limits[i] || q_vec(i) > davinci_kinematics_cuda::q_upper_limits[i]) {
				result -= (1 << i);
			}
		}
		if (result < 0) {
			return result;
		}
		// The wrist is inside the cannula.
		if (q_vec(2) < davinci_kinematics_cuda::cannula_short_length) {
			// if the wrist is straight, then it is ok.
			for (unsigned int i(3); i < 6; i++) {
				if (abs(q_vec(i)) > 0.001) {
					return 1;
				}
			}
		}
		// Test if the gripper is open wider than available.
		if (abs(q_vec(6)) > (davinci_kinematics_cuda::PI - abs(q_vec(5)))) {
			return 2;
		}
		return 0;
	}

	__host__ __device__
	double Forward::dh_var_to_qvec(double dh_val, int index) {
		return (dh_val - davinci_kinematics_cuda::DH_q_offsets[index]);
	}

	// RN TODO deal with that .99 sacle factor
	//    Eigen::VectorXd thetas_DH_vec_,dvals_DH_vec_;
	__host__ __device__
	Vectorq7x1 Forward::convert_DH_vecs_to_qvec(const Eigen::VectorXd &thetas_DH_vec, const Eigen::VectorXd &dvals_DH_vec) {
		Vectorq7x1 q_vec;

		for (int i = 0; i < 7; i++) {
			q_vec(i) = thetas_DH_vec(i) - this->theta_DH_offsets_(i);
		}
		q_vec(2) = dvals_DH_vec(2) - this->dval_DH_offsets_(2);

		return q_vec;
	}

	// given 4 DH parameters, compute the corresponding transform as an affine3d
	__host__ __device__
	Eigen::Affine3d Forward::computeAffineOfDH(double a, double d, double alpha, double theta) {
		Eigen::Affine3d affine_DH;
		Eigen::Matrix3d R;
		Eigen::Vector3d p;

		double cq = cos(theta);
		double sq = sin(theta);
		double sa = sin(alpha);
		double ca = cos(alpha);
		R(0, 0) = cq;

		// - sin(q(i))*cos(alpha);
		R(0, 1) = -sq * ca;

		// sin(q(i))*sin(alpha);
		R(0, 2) = sq * sa;
		R(1, 0) = sq;
		// cos(q(i))*cos(alpha);
		R(1, 1) = cq * ca;
		R(1, 2) = -cq * sa;

		R(2, 0) = 0;
		R(2, 1) = sa;
		R(2, 2) = ca;
		affine_DH.linear() = R;

		p(0) = a * cq;
		p(1) = a * sq;
		p(2) = d;
		affine_DH.translation() = p;

		return affine_DH;
	}

	// provide DH theta and d values, return affine pose of gripper tip w/rt base frame
	// also computes all intermediate affine frames, w/rt base frame
	__host__ __device__
	void Forward::fwd_kin_solve_DH(const Eigen::VectorXd &theta_vec, const Eigen::VectorXd &d_vec) {
		// use or affect these member variables:

		Eigen::Affine3d *affines_i_wrt_iminus1 = (Eigen::Affine3d*) malloc(7 * sizeof(Eigen::Affine3d));
		Eigen::Affine3d transform;
		double a, d, theta, alpha;

		for (int i = 0; i < 7; i++) {
			a = DH_a_params[i];
			d = d_vec(i);
			alpha = DH_alpha_params[i];
			theta = theta_vec(i);
			transform = computeAffineOfDH(a, d, alpha, theta);
			affines_i_wrt_iminus1[i] = transform;
		}

		this->affine_products_ = (Eigen::Affine3d*) malloc(7 * sizeof(double));
		this->affine_products_[0] = this->affine_frame0_wrt_base_ * affines_i_wrt_iminus1[0];
		// RN Note that it starts from 1.
		for (int i = 1; i < 7; i++) {
			this->affine_products_[i] = this->affine_products_[i - 1] * affines_i_wrt_iminus1[i];
		}
		this->affine_gripper_wrt_base_ = this->affine_products_[6] * affine_gripper_wrt_frame6_;

		// RN added for wrist pt coordinate w/rt base frame
		this->affine_wrist_wrt_base_ = this->affine_products_[2];
	}

	__host__ __device__
	Eigen::Affine3d Forward::get_wrist_wrt_base() // RN
	{
		return this->affine_wrist_wrt_base_;
	}
	
	__host__ __device__
	Eigen::Affine3d Forward::fwd_kin_solve(const Vectorq7x1 &q_vec, const unsigned int desired_joint) {
		unsigned int joint = desired_joint - 1;
		this->current_joint_state_ = q_vec;
		Eigen::VectorXd thetas_DH_vec, dvals_DH_vec;
		convert_qvec_to_DH_vecs(q_vec, thetas_DH_vec, dvals_DH_vec);
		fwd_kin_solve_DH(thetas_DH_vec, dvals_DH_vec);

		if (joint < 6) {
			return this->affine_products_[joint];
		} else {
			return this->affine_gripper_wrt_base_;
		}
	}

	__host__ __device__
	Eigen::Affine3d Forward::fwd_kin_solve(const double *q_vec, const unsigned int desired_joint) {
		Vectorq7x1 q;
		q[0] = q_vec[0];
		q[1] = q_vec[1];
		q[2] = q_vec[2];
		q[3] = q_vec[3];
		q[4] = q_vec[4];
		q[5] = q_vec[5];
		q[6] = q_vec[6];
		return this->fwd_kin_solve(q, desired_joint);
	}

	/*
	Eigen::Affine3d Forward::fwd_kin_solve(const sensor_msgs::JointState &jointStateMsg, const unsigned int desiredJoint) {

		//TODO:  Add error checking to ensure jointStateMsg has proper values
		Vectorq7x1 q;

		for (unsigned int i = 0; i < jointStateMsg.name.size(); i++) {
			unsigned int position = davinci_kinematics::jointNameToOrder.find(jointStateMsg.name[i])->second;
			q[position] = jointStateMsg.position[i];
		}

		return this->fwd_kin_solve(q, desiredJoint);
	}*/

	__host__ __device__
	Eigen::Affine3d Forward::fwd_kin_solve() {
		return this->affine_gripper_wrt_base_;
	}

	__host__ __device__
	Eigen::Affine3d Forward::get_frame0_wrt_base() const {
		return this->affine_frame0_wrt_base_;
	}

	__host__ __device__
	void Forward::set_frame0_wrt_base(const Eigen::Affine3d &affine_frame0_wrt_base) {
		this->affine_frame0_wrt_base_ = affine_frame0_wrt_base;
	}

	__host__ __device__
	Eigen::Affine3d Forward::get_gripper_wrt_frame6() const {
		return this->affine_gripper_wrt_frame6_;
	}

	__host__ __device__
	void Forward::set_gripper_jaw_length(double jaw_length) {
		this->gripper_jaw_length_ = jaw_length;
		this->affine_gripper_wrt_frame6_ = computeAffineOfDH(0, this->gripper_jaw_length_, 0, -davinci_kinematics_cuda::PI / 2);
	}

	__host__ __device__
	Eigen::MatrixXd Forward::compute_jacobian(const Vectorq7x1 &q_vec) {
		// use the jacobian to make the computation.
		fwd_kin_solve(q_vec);
		Eigen::Vector3d z_axis;
		Eigen::Vector3d vec_tip_minus_Oi_wrt_base;
		Eigen::Matrix3d R;
		Eigen::Vector3d r_tip_wrt_base = this->affine_gripper_wrt_base_.translation();
		Eigen::Vector3d z_axis0 = this->affine_frame0_wrt_base_.linear().col(2);
		// angular Jacobian is just the z axes of each revolute joint (expressed in base frame);
		// for prismatic joint, there is no angular contribution
		// start from z_axis0

		// Block of size (p,q), starting at (i,j) matrix.block<p,q>(i,j);
		this->Jacobian_.block<3, 1>(3, 0) = z_axis0;
		vec_tip_minus_Oi_wrt_base = r_tip_wrt_base - this->affine_frame0_wrt_base_.translation();
		this->Jacobian_.block<3, 1>(0, 0) = z_axis0.cross(vec_tip_minus_Oi_wrt_base);
		// 2nd joint:
		// refer to previous joint's z axis
		R = this->affine_products_[0].linear();
		z_axis = R.col(2);
		// Block of size (p,q), starting at (i,j) matrix.block<p,q>(i,j);
		this->Jacobian_.block<3, 1>(3, 1) = z_axis;
		vec_tip_minus_Oi_wrt_base = r_tip_wrt_base - this->affine_products_[0].translation();
		this->Jacobian_.block<3, 1>(0, 1) = z_axis.cross(vec_tip_minus_Oi_wrt_base);

		// prismatic joint:
		R = this->affine_products_[1].linear();
		z_axis = R.col(2);
		this->Jacobian_.block<3, 1>(0, 2) = z_axis;

		// joints 4-6:
		for (int i = 3; i < 6; i++) {
			R = this->affine_products_[i - 1].linear();
			z_axis = R.col(2);
			// Block of size (p,q), starting at (i,j) matrix.block<p,q>(i,j);
			this->Jacobian_.block<3, 1>(3, i) = z_axis;
			vec_tip_minus_Oi_wrt_base = r_tip_wrt_base - this->affine_products_[i - 1].translation();
			this->Jacobian_.block<3, 1>(0, i) = z_axis.cross(vec_tip_minus_Oi_wrt_base);
		}
		// translational Jacobian depends on joint's z-axis and vector from i'th axis to robot tip
		return this->Jacobian_;
	}

	__host__ __device__
	Eigen::MatrixXd Forward::compute_jacobian() {
		return this->Jacobian_;
	}

	__host__ __device__
	void Forward::resetDhGenericParams() {
		this->theta_DH_offsets_generic_.resize(7);
		this->dval_DH_offsets_generic_.resize(7);
		this->DH_a_params_generic_.resize(7);
		this->DH_alpha_params_generic_.resize(7);

		for (int i = 0; i < 7; i++) {
			this->theta_DH_offsets_generic_(i) = davinci_kinematics_cuda::DH_q_offsets[i];
			this->DH_alpha_params_generic_(i) = davinci_kinematics_cuda::DH_alpha_params[i];
			this->DH_a_params_generic_(i) = davinci_kinematics_cuda::DH_a_params[i];
			if (i == 2) {
				this->theta_DH_offsets_generic_(i) = 0;
			}
		}

		this->dval_DH_offsets_generic_ << 0, 0, davinci_kinematics_cuda::DH_q_offsets[2], 0, 0, 0, 0;
		this->j1_scale_factor_generic_ = 1.0;
		this->j2_scale_factor_generic_ = 1.0;
		this->j3_scale_factor_generic_ = 1.0;
	}
}  // namespace davinci_kinematics
