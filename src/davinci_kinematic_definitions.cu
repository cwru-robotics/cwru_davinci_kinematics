/*
 *  davinci_kinematic_definitions.cuh
 *  Copyright (C) 2017  Wyatt S. Newman, Russell C. Jackson, and Tom Shkurti.
 *  Copyright (C) 2021 Ethan Shafer.
 *  
 *  This program is free software: you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation, either version 3 of the License, or
 *  (at your option) any later version.

 *  This program is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.

 *  You should have received a copy of the GNU General Public License
 *  along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <davinci_kinematics_cuda/davinci_kinematic_definitions.cuh>

namespace davinci_kinematics_cuda {
	// the long and short distance from the portal to the exit of the cannula.

}  // namespace davinci_kinematics
